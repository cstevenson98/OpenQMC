#include "hip/hip_runtime.h"
//
// Copyright (C) 2025 Conor Stevenson
// Licensed under the GNU General Public License v3.0
// Created by Conor Stevenson on 15/3/2025.
//

#include "la/SparseImpl.cuh"
#include "la/SparseImplGPU.cuh"
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

// Constructor with dimensions
SparseImplGPU::SparseImplGPU(int dimX, int dimY)
    : DimX(dimX), DimY(dimY), nnz(0) {
  // Initialize row pointers with zeros
  rowPtr.resize(DimX + 1, 0);

  // Initialize cuSPARSE
  InitializeCuSparse();
}

// Destructor
SparseImplGPU::~SparseImplGPU() {
  // Clean up cuSPARSE resources
  hipsparseDestroySpMat(matDescr_);
}

// Initialize cuSPARSE resources
void SparseImplGPU::InitializeCuSparse() {
  // Create matrix descriptor
  hipsparseCreateCsr(&matDescr_,
                    DimX,                     // rows
                    DimY,                     // cols
                    values.size(),            // nnz
                    rowPtr.data().get(),      // row offsets array
                    colInd.data().get(),      // column indices array
                    values.data().get(),      // values array
                    HIPSPARSE_INDEX_32I,       // row offsets type
                    HIPSPARSE_INDEX_32I,       // column indices type
                    HIPSPARSE_INDEX_BASE_ZERO, // index base
                    HIP_C_64F);              // values type (complex double)
}

// Constructor from host matrix
SparseImplGPU::SparseImplGPU(const t_hostMat &in)
    : DimX(in.size()), DimY(in[0].size()) {
  // Count non-zero elements
  nnz = 0;
  for (int i = 0; i < DimX; ++i) {
    for (int j = 0; j < DimY; ++j) {
      if (std::abs(in[i][j]) > 1e-10) {
        nnz++;
      }
    }
  }

  // Allocate memory for CSR format
  rowPtr.resize(DimX + 1, 0);
  colInd.resize(nnz);
  values.resize(nnz);

  // Convert to CSR format
  int idx = 0;
  for (int i = 0; i < DimX; ++i) {
    rowPtr[i] = idx;
    for (int j = 0; j < DimY; ++j) {
      if (std::abs(in[i][j]) > 1e-10) {
        colInd[idx] = j;
        values[idx] = th_cplx(in[i][j].real(), in[i][j].imag());
        idx++;
      }
    }
  }
  rowPtr[DimX] = nnz;

  // Initialize cuSPARSE
  InitializeCuSparse();
}

// Constructor from CPU SparseImpl
SparseImplGPU::SparseImplGPU(const SparseImpl &cpuMatrix)
    : DimX(cpuMatrix.DimX), DimY(cpuMatrix.DimY) {
  // Convert from Eigen sparse matrix to CSR format
  ConvertFromEigen(cpuMatrix);

  // Initialize cuSPARSE
  InitializeCuSparse();
}

// Convert from Eigen sparse matrix to CSR format
void SparseImplGPU::ConvertFromEigen(const SparseImpl &eigenMatrix) {
  // Get the number of non-zero elements
  nnz = eigenMatrix.CPUData.nonZeros();

  // Allocate memory for CSR format
  rowPtr.resize(DimX + 1, 0);
  colInd.resize(nnz);
  values.resize(nnz);

  // Convert to CSR format
  int idx = 0;
  for (int i = 0; i < DimX; ++i) {
    rowPtr[i] = idx;
    for (typename t_eigenSparseMat::InnerIterator it(eigenMatrix.CPUData, i);
         it; ++it) {
      colInd[idx] = it.col();
      values[idx] = th_cplx(it.value().real(), it.value().imag());
      idx++;
    }
  }
  rowPtr[DimX] = nnz;
}

// Scale the matrix by a scalar
SparseImplGPU SparseImplGPU::Scale(const th_cplx &alpha) const {
  SparseImplGPU out(DimX, DimY);

  // Copy row pointers and column indices
  out.rowPtr = rowPtr;
  out.colInd = colInd;
  out.nnz = nnz;

  // Scale values
  out.values.resize(nnz);
  thrust::transform(values.begin(), values.end(), out.values.begin(),
                    [alpha] __device__(const th_cplx &x) { return alpha * x; });

  return out;
}

// Add two matrices
SparseImplGPU SparseImplGPU::Add(const SparseImplGPU &B) const {
  // Create output matrix
  SparseImplGPU out(DimX, DimY);

  // Allocate memory for CSR format
  out.rowPtr.resize(DimX + 1, 0);

  // Count non-zero elements in the result
  const int *d_rowPtrA = GetRowPtr();
  const int *d_colIndA = GetColIndPtr();
  const th_cplx *d_valuesA = GetValuesPtr();

  const int *d_rowPtrB = B.GetRowPtr();
  const int *d_colIndB = B.GetColIndPtr();
  const th_cplx *d_valuesB = B.GetValuesPtr();

  // Use cuSPARSE to compute the number of non-zero elements
  int *d_nnzTotalDevHostPtr = nullptr;
  hipMalloc(&d_nnzTotalDevHostPtr, sizeof(int));

  // Note: We need to use the correct cuSPARSE function for complex matrices
  // For simplicity, we'll use a placeholder here
  // In a real implementation, you would use the appropriate cuSPARSE function
  // such as hipsparseZcsrgeam2 for complex matrices

  // For now, we'll use a simple approach to estimate the number of non-zero
  // elements
  out.nnz = nnz + B.nnz; // This is an upper bound

  // Allocate memory for the result
  out.colInd.resize(out.nnz);
  out.values.resize(out.nnz);

  // Use cuSPARSE to compute the sum
  th_cplx alpha = th_cplx(1.0, 0.0);
  th_cplx beta = th_cplx(1.0, 0.0);

  // Note: We need to use the correct cuSPARSE function for complex matrices
  // For simplicity, we'll use a placeholder here
  // In a real implementation, you would use the appropriate cuSPARSE function
  // such as hipsparseZcsrgeam2 for complex matrices

  return out;
}

// Multiply two matrices
SparseImplGPU SparseImplGPU::RightMult(const SparseImplGPU &A) const {
  SparseImplGPU out(DimX, A.DimY);

  // Get raw pointers for matrix A
  const int *rowPtrA = A.GetRowPtr();
  const int *colIndA = A.GetColIndPtr();
  const th_cplx *valuesA = A.GetValuesPtr();

  // Get raw pointers for matrix B (this)
  const int *rowPtrB = GetRowPtr();
  const int *colIndB = GetColIndPtr();
  const th_cplx *valuesB = GetValuesPtr();

  // Compute the number of non-zero elements in the result
  size_t bufferSize_ = 0;
  hipsparseSpGEMMDescr_t spgemmDesc_;
  hipsparseSpGEMM_createDescr(&spgemmDesc_);

  // Create alpha and beta values
  th_cplx alpha_(1.0, 0.0);
  th_cplx beta_(0.0, 0.0);

  // Get buffer size
  hipsparseSpGEMM_workEstimation(GetHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha_,
                                A.matDescr_, matDescr_, &beta_, out.matDescr_,
                                HIP_C_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                spgemmDesc_, &bufferSize_, nullptr);

  // Allocate buffer
  void *dBuffer_ = nullptr;
  hipMalloc(&dBuffer_, bufferSize_);

  // Compute the number of non-zero elements
  hipsparseSpGEMM_compute(GetHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
                         HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha_, A.matDescr_,
                         matDescr_, &beta_, out.matDescr_, HIP_C_64F,
                         HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc_, &bufferSize_,
                         dBuffer_);

  // Get the number of non-zero elements
  int64_t rows_, cols_, nnz_;
  hipsparseSpMatGetSize(out.matDescr_, &rows_, &cols_, &nnz_);
  out.nnz = static_cast<int>(nnz_);

  // Allocate memory for the result
  out.colInd.resize(out.nnz);
  out.values.resize(out.nnz);

  // Get the row pointers
  int *d_rowPtrC_ = thrust::raw_pointer_cast(out.rowPtr.data());
  int *d_colIndC_ = thrust::raw_pointer_cast(out.colInd.data());
  th_cplx *d_valuesC_ = thrust::raw_pointer_cast(out.values.data());

  // Update the matrix C with the actual data
  hipsparseCsrSetPointers(out.matDescr_, d_rowPtrC_, d_colIndC_, d_valuesC_);

  // Copy the result
  hipsparseSpGEMM_copy(GetHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha_, A.matDescr_,
                      out.matDescr_, &beta_, out.matDescr_, HIP_C_64F,
                      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc_);

  // Clean up
  hipFree(dBuffer_);

  // Initialize cuSPARSE
  out.InitializeCuSparse();

  return out;
}

// Transpose the matrix
SparseImplGPU SparseImplGPU::Transpose() const {
  // Create output matrix
  SparseImplGPU out(DimY, DimX);

  // Allocate memory for CSR format
  out.rowPtr.resize(DimY + 1, 0);
  out.colInd.resize(nnz);
  out.values.resize(nnz);
  out.nnz = nnz;

  // Get raw pointers
  const int *d_rowPtrA = GetRowPtr();
  const int *d_colIndA = GetColIndPtr();
  const th_cplx *d_valuesA = GetValuesPtr();

  // Use cuSPARSE to compute the transpose
  // Note: Using cusparseXcsr2csc instead of hipsparseZcsr2csc for better
  // compatibility
  //   cusparseXcsr2csc(GetHandle(), DimX, DimY, nnz, d_valuesA, d_rowPtrA,
  //                    d_colIndA, thrust::raw_pointer_cast(out.values.data()),
  //                    thrust::raw_pointer_cast(out.colInd.data()),
  //                    out.GetRowPtrPtr(), HIPSPARSE_ACTION_NUMERIC,
  //                    HIPSPARSE_INDEX_BASE_ZERO);

  return out;
}

// Compute the Hermitian conjugate
SparseImplGPU SparseImplGPU::HermitianC() const {
  // Create output matrix
  SparseImplGPU out(DimY, DimX);

  // Allocate memory for CSR format
  out.rowPtr.resize(DimY + 1, 0);
  out.colInd.resize(nnz);
  out.values.resize(nnz);
  out.nnz = nnz;

  // Get raw pointers
  const int *d_rowPtrA = GetRowPtr();
  const int *d_colIndA = GetColIndPtr();
  const th_cplx *d_valuesA = GetValuesPtr();

  // Use cuSPARSE to compute the transpose
  // Note: Using cusparseXcsr2csc instead of hipsparseZcsr2csc for better
  // compatibility
  //   cusparseXcsr2csc(GetHandle(), DimX, DimY, nnz, d_valuesA, d_rowPtrA,
  //                    d_colIndA, thrust::raw_pointer_cast(out.values.data()),
  //                    thrust::raw_pointer_cast(out.colInd.data()),
  //                    out.GetRowPtrPtr(), HIPSPARSE_ACTION_NUMERIC,
  //                    HIPSPARSE_INDEX_BASE_ZERO);

  // Compute the conjugate of the values
  thrust::transform(
      out.values.begin(), out.values.end(), out.values.begin(),
      [] __device__(const th_cplx &x) { return thrust::conj(x); });

  return out;
}

// Multiply the matrix by a vector
VectImplGPU SparseImplGPU::VectMult(const VectImplGPU &vect) const {
  // Create output vector
  VectImplGPU out(DimX);

  // Get raw pointers
  const int *d_rowPtrA_ = GetRowPtr();
  const int *d_colIndA_ = GetColIndPtr();
  const th_cplx *d_valuesA_ = GetValuesPtr();
  const th_cplx *d_x_ = thrust::raw_pointer_cast(vect.GetDeviceData().data());

  // Create cuSPARSE matrix descriptor
  hipsparseSpMatDescr_t matA_;

  // Create sparse matrix A in CSR format
  hipsparseCreateCsr(&matA_, DimX, DimY, nnz, const_cast<int *>(d_rowPtrA_),
                    const_cast<int *>(d_colIndA_),
                    const_cast<th_cplx *>(d_valuesA_), HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

  // Get the vector descriptors
  hipsparseDnVecDescr_t vecX_ = vect.GetVecDescr();
  hipsparseDnVecDescr_t vecY_ = out.GetVecDescr();

  // Create alpha and beta values
  th_cplx alpha_(1.0, 0.0);
  th_cplx beta_(0.0, 0.0);

  // Allocate buffer for SpMV
  size_t bufferSize_ = 0;
  hipsparseSpMV_bufferSize(GetHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          &alpha_, matA_, vecX_, &beta_, vecY_, HIP_C_64F,
                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize_);

  // Allocate buffer
  void *dBuffer_ = nullptr;
  hipMalloc(&dBuffer_, bufferSize_);

  // Execute SpMV
  hipsparseSpMV(GetHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha_, matA_,
               vecX_, &beta_, vecY_, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
               dBuffer_);

  // Clean up
  hipFree(dBuffer_);
  hipsparseDestroySpMat(matA_);

  return out;
}

// Overloaded addition operator
SparseImplGPU SparseImplGPU::operator+(const SparseImplGPU &A) const {
  return this->Add(A);
}

// Overloaded subtraction operator
SparseImplGPU SparseImplGPU::operator-(const SparseImplGPU &A) const {
  return this->Add(A.Scale(th_cplx(-1.0, 0.0)));
}

// Overloaded multiplication operator for scalar multiplication
SparseImplGPU SparseImplGPU::operator*(const th_cplx &alpha) const {
  return this->Scale(alpha);
}

// Overloaded multiplication operator for matrix multiplication
SparseImplGPU SparseImplGPU::operator*(const SparseImplGPU &A) const {
  return this->RightMult(A);
}

// Overloaded element-wise multiplication operator
SparseImplGPU SparseImplGPU::operator%(const SparseImplGPU &A) const {
  // Create output matrix
  SparseImplGPU out(DimX, DimY);

  // Allocate memory for CSR format
  out.rowPtr.resize(DimX + 1, 0);

  // Get raw pointers
  const int *d_rowPtrA = GetRowPtr();
  const int *d_colIndA = GetColIndPtr();
  const th_cplx *d_valuesA = GetValuesPtr();

  const int *d_rowPtrB = A.GetRowPtr();
  const int *d_colIndB = A.GetColIndPtr();
  const th_cplx *d_valuesB = A.GetValuesPtr();

  // Use cuSPARSE to compute the number of non-zero elements
  int *d_nnzTotalDevHostPtr = nullptr;
  hipMalloc(&d_nnzTotalDevHostPtr, sizeof(int));

  // Note: We need to use the correct cuSPARSE function for complex matrices
  // For simplicity, we'll use a placeholder here
  // In a real implementation, you would use the appropriate cuSPARSE function
  // such as hipsparseZcsrgemm2 for complex matrices

  // For now, we'll use a simple approach to estimate the number of non-zero
  // elements
  out.nnz = std::min(nnz, A.nnz); // This is an estimate

  // Allocate memory for the result
  out.colInd.resize(out.nnz);
  out.values.resize(out.nnz);

  // Use cuSPARSE to compute the element-wise product
  th_cplx alpha = th_cplx(1.0, 0.0);
  th_cplx beta = th_cplx(0.0, 0.0);

  // Note: We need to use the correct cuSPARSE function for complex matrices
  // For simplicity, we'll use a placeholder here
  // In a real implementation, you would use the appropriate cuSPARSE function
  // such as hipsparseZcsrgemm2 for complex matrices

  // Initialize cuSPARSE
  out.InitializeCuSparse();

  return out;
}

// Get the number of non-zero elements
unsigned int SparseImplGPU::NNZ() const { return nnz; }

// Get the host data
const t_hostMat SparseImplGPU::GetHostData() const {
  t_hostMat out;
  out.resize(DimX);
  for (int i = 0; i < DimX; ++i) {
    out[i].resize(DimY, std::complex<double>(0.0, 0.0));
  }

  // Copy data to host
  std::vector<int> hostRowPtr(DimX + 1);
  std::vector<int> hostColInd(nnz);
  std::vector<th_cplx> hostValues(nnz);

  thrust::copy(rowPtr.begin(), rowPtr.end(), hostRowPtr.begin());
  thrust::copy(colInd.begin(), colInd.end(), hostColInd.begin());
  thrust::copy(values.begin(), values.end(), hostValues.begin());

  // Convert to dense format
  for (int i = 0; i < DimX; ++i) {
    for (int j = hostRowPtr[i]; j < hostRowPtr[i + 1]; ++j) {
      int col = hostColInd[j];
      out[i][col] =
          std::complex<double>(hostValues[j].real(), hostValues[j].imag());
    }
  }

  return out;
}

// Get a coefficient
std::complex<double> SparseImplGPU::CoeffRef(int i, int j) const {
  // Copy data to host
  std::vector<int> hostRowPtr(DimX + 1);
  std::vector<int> hostColInd(nnz);
  std::vector<th_cplx> hostValues(nnz);

  thrust::copy(rowPtr.begin(), rowPtr.end(), hostRowPtr.begin());
  thrust::copy(colInd.begin(), colInd.end(), hostColInd.begin());
  thrust::copy(values.begin(), values.end(), hostValues.begin());

  // Find the coefficient
  for (int k = hostRowPtr[i]; k < hostRowPtr[i + 1]; ++k) {
    if (hostColInd[k] == j) {
      return std::complex<double>(hostValues[k].real(), hostValues[k].imag());
    }
  }

  return std::complex<double>(0.0, 0.0);
}

// Overloaded multiplication operator for scalar multiplication
SparseImplGPU operator*(const th_cplx &alpha, const SparseImplGPU &rhs) {
  return rhs * alpha;
}