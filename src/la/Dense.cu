#include "hip/hip_runtime.h"
//
// Copyright (C) 2025 Conor Stevenson
// Licensed under the GNU General Public License v3.0
// Created by Conor Stevenson on 03/04/2022.
//

#include "core/types.cuh"
#include "core/types.h"
#include "la/Dense.h"
#include "la/DenseImpl.cuh"

#include <cassert>
#include <complex>
#include <iostream>
#include <memory>
#include <vector>

// DenseImpl constructor
DenseImpl::DenseImpl() noexcept : DimX(0), DimY(0) {}

// DenseImpl destructor
DenseImpl::~DenseImpl() noexcept = default;

// DenseImpl constructor
DenseImpl::DenseImpl(int dimX, int dimY) : DimX(dimX), DimY(dimY) {
  if (DimX < 0 || DimY < 0) {
    throw std::invalid_argument("Invalid dimensions for Dense matrix.");
  }
  CPUData.resize(DimX);
  for (int i = 0; i < DimX; ++i) {
    CPUData[i].resize(DimY);
  }
}

// DenseImpl copy constructor
DenseImpl::DenseImpl(const DenseImpl &other) noexcept
    : DimX(other.DimX), DimY(other.DimY), CPUData(other.CPUData) {}

// DenseImpl move constructor
DenseImpl::DenseImpl(DenseImpl &&other) noexcept
    : DimX(other.DimX), DimY(other.DimY), CPUData(std::move(other.CPUData)) {}

// DenseImpl constructor
DenseImpl::DenseImpl(t_hostMat &in) noexcept
    : DimX(in.size()), DimY(in[0].size()) {
  CPUData = in;
}

// Copy assignment operator
DenseImpl &DenseImpl::operator=(const DenseImpl &other) noexcept {
  if (this == &other) {
    return *this;
  }

  DimX = other.DimX;
  DimY = other.DimY;
  CPUData = other.CPUData;

  return *this;
}

DenseImpl DenseImpl::Add(const DenseImpl &A) const {
  // if dimensions don't match, throw
  if (DimX != A.DimX || DimY != A.DimY) {
    throw std::invalid_argument("Dimensions do not match for Dense matrix.");
  }

  DenseImpl out(DimX, DimY);
  for (int i = 0; i < DimX; ++i) {
    for (int j = 0; j < DimY; ++j) {
      out.CPUData[i][j] = CPUData[i][j] + A.CPUData[i][j];
    }
  }
  return out;
}

DenseImpl DenseImpl::RightMult(const DenseImpl &A) const {
  // if dimensions don't match, throw
  if (DimY != A.DimX) {
    throw std::invalid_argument("Dimensions do not match for Dense matrix.");
  }

  DenseImpl out(DimX, A.DimY);
  for (int i = 0; i < DimX; ++i) {
    for (int j = 0; j < DimY; ++j) {
      t_cplx sum = 0;
      for (int k = 0; k < DimY; ++k) {
        sum += CPUData[i][k] * A.CPUData[k][j];
      }
      out.CPUData[i][j] = sum;
    }
  }

  return out;
}

DenseImpl DenseImpl::Scale(t_cplx alpha) const noexcept {
  DenseImpl out(DimX, DimY);

  for (int i = 0; i < out.CPUData.size(); ++i) {
    for (int j = 0; j < out.CPUData[0].size(); ++j) {
      out.CPUData[i][j] = alpha * CPUData[i][j];
    }
  }

  return out;
}

DenseImpl DenseImpl::Transpose() const noexcept {
  DenseImpl out(DimY, DimX);

  for (int i = 0; i < DimY; ++i) {
    for (int j = 0; j < DimX; ++j) {
      out.CPUData[i][j] = CPUData[j][i];
    }
  }

  return out;
}

DenseImpl DenseImpl::HermitianC() const noexcept {
  DenseImpl out(DimY, DimX);

  for (int i = 0; i < DimY; ++i) {
    for (int j = 0; j < DimX; ++j) {
      out.CPUData[i][j] = conj(CPUData[j][i]);
    }
  }

  return out;
}

t_hostVect DenseImpl::FlattenedData() const noexcept {
  t_hostVect out;
  out.resize(DimX * DimY);

  for (int i = 0; i < DimX; i++) {
    for (int j = 0; j < DimY; j++) {
      out[j + i * DimY] = CPUData[i][j];
    }
  }

  return out;
}

void DenseImpl::Print(unsigned int kind, unsigned int prec) const noexcept {
  std::string s;
  std::stringstream stream;
  stream.setf(std::ios::fixed);
  stream.precision(prec);

  stream << " Matrix [" << DimX << " x " << DimY << "]:" << std::endl;
  for (const auto &X : CPUData) {
    stream << "   ";
    for (auto Y : X) {
      std::string spaceCharRe = !std::signbit(Y.real()) ? " " : "";
      std::string spaceCharIm = !std::signbit(Y.imag()) ? " " : "";
      std::string spaceCharAbs = !std::signbit(Y.imag()) ? " + " : "-";

      switch (kind) {
      case 0: // re + im
        stream << spaceCharRe << Y.real() << spaceCharAbs << abs(Y.imag())
               << "i  ";
        break;
      case 1: // re
        stream << spaceCharRe << Y.real() << " ";
        break;
      case 2: // im
        stream << spaceCharIm << Y.imag() << "i  ";
        break;
      case 3: // abs
        stream << " " << abs(Y);
        break;
      default:
        stream << "[e]";
      }
    }
    stream << std::endl;
  }

  s = stream.str();

  std::cout << s << std::endl;
}

////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////
// Definition of Dense
////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////

// Default constructor
Dense::Dense() noexcept : pImpl(std::make_unique<DenseImpl>()) {}

// Dense constructor
Dense::Dense(int dimX, int dimY)
    : pImpl(std::make_unique<DenseImpl>(dimX, dimY)) {
  // throw if dimensions are invalid
  if (dimX < 0 || dimY < 0) {
    throw std::invalid_argument("Invalid dimensions for Dense matrix.");
  }
}

// Dense constructor
Dense::Dense(t_hostMat &in) noexcept : pImpl(std::make_unique<DenseImpl>(in)) {
  pImpl->CPUData = in;
}

// Destructor
Dense::~Dense() noexcept = default;

/**
 * @brief Dense copy constructor
 */
Dense::Dense(const Dense &other) noexcept
    : pImpl(std::make_unique<DenseImpl>(*other.pImpl)) {}

/*
 * @brief Dense move constructor
 */
Dense::Dense(Dense &&other) noexcept : pImpl(std::move(other.pImpl)) {}

/**
 * @brief Dense non-empty constructor
 */
Dense::Dense(std::unique_ptr<DenseImpl> impl) noexcept
    : pImpl(std::move(impl)) {}

// Copy assignment operator
Dense &Dense::operator=(const Dense &other) noexcept {
  if (this == &other) {
    return *this;
  }

  pImpl = std::make_unique<DenseImpl>(*other.pImpl);

  return *this;
}

/**
 * @brief Get the number of rows in the Dense matrix.
 *
 * @return int Number of rows.
 */

int Dense::DimX() const { return pImpl->DimX; }

/**
 * @brief Get the number of columns in the Dense matrix.
 *
 * @return int Number of columns.
 */
int Dense::DimY() const { return pImpl->DimY; }

const t_hostMat &Dense::GetHostData() const { return pImpl->CPUData; }

/**
 * @brief Get the data at a specific position in the Dense matrix.
 *
 * @param i Row index.
 * @param j Column index.
 * @return std::complex<double> Element at the specified position.
 */
std::complex<double> &Dense::GetData(int i, int j) const {
  // if out of bounds, throw
  if (i < 0 || i >= pImpl->DimX || j < 0 || j >= pImpl->DimY) {
    throw std::out_of_range("Index out of bounds for Dense matrix.");
  }
  return pImpl->CPUData[i][j];
}

// at access, which will never throw
std::complex<double> Dense::at(int i, int j) const noexcept {
  // make sure in range
  if (i < 0 || i >= pImpl->DimX || j < 0 || j >= pImpl->DimY) {
    return {-1337., -1337.};
  }
  return pImpl->CPUData[i][j];
}

/**
 * @brief Get a reference to the data at a specific position in the Dense
 * matrix.
 *
 * @param i Row index.
 * @param j Column index.
 * @return std::complex<double>& Reference to the element at the specified
 * position.
 */
std::complex<double> &Dense::GetDataRef(int i, int j) const {
  return pImpl->CPUData[i][j];
}

/**
 * @brief Overloaded subscript operator to access matrix elements.
 *
 * @param col Column index.
 * @param row Row index.
 * @return std::complex<double> Element at the specified position.
 */
std::complex<double> Dense::operator[](int col, int row) {
  if (pImpl != nullptr) {
    return pImpl->CPUData[col][row];
  }
  return -1337.;
}

/**
 * @brief Move assignment operator.
 *
 * @param other Another Dense object to move from.
 * @return Dense& Reference to the current object.
 */
Dense &Dense::operator=(Dense &&other) noexcept {
  if (this == &other) {
    return *this;
  }

  pImpl = std::move(other.pImpl);

  return *this;
}

/**
 * @brief Overloaded addition operator for Dense matrices.
 *
 * @param A Another Dense object to add.
 * @return Dense Result of the addition.
 */
Dense Dense::operator+(const Dense &A) const {
  return Dense(std::make_unique<DenseImpl>(pImpl->Add(*A.pImpl)));
}

/**
 * @brief Overloaded multiplication operator for Dense matrices.
 *
 * @param A Another Dense object to multiply.
 * @return Dense Result of the multiplication.
 */
Dense Dense::operator*(const Dense &A) const {
  return Dense(std::make_unique<DenseImpl>(pImpl->RightMult(*A.pImpl)));
}

/**
 * @brief Overloaded subtraction operator for Dense matrices.
 *
 * @param A Another Dense object to subtract.
 * @return Dense Result of the subtraction.
 */
Dense Dense::operator-(const Dense &A) const {
  return this->operator+(A.operator*(t_cplx(-1)));
}

/**
 * @brief Overloaded multiplication operator for scalar multiplication.
 *
 * @param alpha Scalar value to multiply.
 * @return Dense Result of the scalar multiplication.
 */
Dense Dense::operator*(const t_cplx &alpha) const noexcept {
  return Dense(std::make_unique<DenseImpl>(pImpl->Scale(alpha)));
}

/**
 * @brief Overloaded multiplication operator for scalar multiplication.
 *
 * @param alpha Scalar value to multiply.
 * @param rhs Dense object to multiply.
 * @return Dense Result of the scalar multiplication.
 */
Dense operator*(const t_cplx &alpha, const Dense &rhs) noexcept {
  return rhs * alpha;
}

/**
 * @brief Transpose the Dense matrix.
 *
 * @return Dense Transposed matrix.
 */
Dense Dense::Transpose() const noexcept {
  return Dense(std::make_unique<DenseImpl>(pImpl->Transpose()));
}

/**
 * @brief Compute the Hermitian conjugate of the Dense matrix.
 *
 * @return Dense Hermitian conjugate matrix.
 */
Dense Dense::HermitianC() const noexcept {
  return Dense(std::make_unique<DenseImpl>(pImpl->HermitianC()));
}

/**
 * @brief Flatten the Dense matrix data into a vector.
 *
 * @return t_hostVect Flattened data.
 */
t_hostVect Dense::FlattenedData() const noexcept {
  return pImpl->FlattenedData();
}

/**
 * @brief Print the Dense matrix.
 *
 * @param kind Type of data to print (real, imaginary, etc.).
 * @param prec Precision of the printed data.
 */
void Dense::Print(unsigned int kind, unsigned int prec) const {
  pImpl->Print(kind, prec);
}
