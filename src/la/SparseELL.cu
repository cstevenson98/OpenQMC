#include "hip/hip_runtime.h"
//
// Copyright (C) 2025 Conor Stevenson
// Licensed under the GNU General Public License v3.0
// Created by conor on 11/04/2022.
//

#include "la/Sparse.cuh"
#include "la/SparseELL.cuh"

using t_cplx = thrust::complex<double>;
using t_hostVect = thrust::host_vector<thrust::complex<double>>;

SparseELL ToSparseELL(const Sparse& A) {
    auto rows = SparseRowsCOO(A);

    // determine length of longest row
    unsigned int highestCount = 0;
    for (auto row : rows) {
        unsigned int len = row.RowData.size();
        if (len > highestCount) {
            highestCount = len;
        }
    }

    SparseELL out(A.DimX, A.DimY, highestCount);
    for (auto row : rows) {
        for (int i = 0; i < row.RowData.size(); i++)
        {
            out.Values.Data[row.Index][i] = row.RowData[i].Val;
            out.Indices.Data[row.Index][i] = row.RowData[i].Coords[1];
        }

        // padding with '-1'
        for (int i = row.RowData.size(); i < highestCount; i++) {
            out.Values.Data[row.Index][i] = 0;
            out.Indices.Data[row.Index][i] = -1;
        }
    }

    return out;
}

Vect SparseELL::VectMult(const Vect &vect) const {
    Vect out(vect.Data.size());

    for (int row = 0; row < vect.Data.size(); ++row) {
        out[row] = 0;
        for (int i = 0; i < EntriesPerRow; i++) {
            int    col = floor(Indices.Data[row][i].real());
            t_cplx val = Values.Data[row][i];

            if (col > -1)
                out.Data[row] += val * vect.Data[col];
        }
    }
    return out;
}
