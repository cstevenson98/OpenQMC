#include "hip/hip_runtime.h"
//
// Created by conor on 11/04/2022.
//

#include "SparseELL.cuh"

SparseELL ToSparseELL(const Sparse& A) {
    auto rows = SparseRowsCOO(A);

    // determine length of longest row
    unsigned int highestCount = 0;
    for (auto row : rows) {
        unsigned int len = row.RowData.size();
        if (len > highestCount) {
            highestCount = len;
        }
    }

    SparseELL out(A.DimX, A.DimY, highestCount);
    for (auto row : rows) {
        for (int i = 0; i < row.RowData.size(); i++)
        {
            out.Values.Data[row.Index][i] = row.RowData[i].Val;
            out.Indices.Data[row.Index][i] = row.RowData[i].Coords[1];
        }

        // padding with '-1'
        for (int i = row.RowData.size(); i < highestCount; i++) {
            out.Values.Data[row.Index][i] = 0;
            out.Indices.Data[row.Index][i] = -1;
        }
    }

    return out;
}

Vect SparseELL::VectMult(const Vect &vect) const {
    Vect out(vect.Data.size());

    for (int row = 0; row < vect.Data.size(); ++row) {
        out[row] = 0;
        for (int i = 0; i < EntriesPerRow; i++) {
            int                  col = floor(Indices.Data[row][i].real());
            std::complex<double> val = Values.Data[row][i];

            if (col > -1)
                out.Data[row] += val * vect.Data[col];
        }
    }
    return out;
}
