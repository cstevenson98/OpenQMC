#include "hip/hip_runtime.h"
//
// Copyright (C) 2025 Conor Stevenson
// Licensed under the GNU General Public License v3.0
// Created by conor on 11/04/2022.
//

#include "core/types.cuh"
#include "la/Sparse.cuh"
#include "la/SparseELL.cuh"

SparseELL ToSparseELL(const Sparse &A) {
  auto rows = SparseRowsCOO(A);

  // determine length of longest row
  unsigned int highestCount = 0;
  for (auto row : rows) {
    unsigned int len = row.RowData.size();
    if (len > highestCount) {
      highestCount = len;
    }
  }

  SparseELL out(A.DimX, A.DimY, highestCount);
  for (auto row : rows) {
    for (int i = 0; i < row.RowData.size(); i++) {
      // Change to denseimpls
      // out.Values.GetDataRef(row.Index, i) = row.RowData[i].Val;
      // out.Indices.GetDataRef(row.Index, i) = row.RowData[i].Coords[1];
    }

    // padding with '-1'
    for (int i = row.RowData.size(); i < highestCount; i++) {
      // out.Values.GetDataRef(row.Index, i) = 0;
      // out.Indices.GetDataRef(row.Index, i) = -1;
    }
  }

  return out;
}

// Vect SparseELL::VectMult(const Vect::Impl &vect) const {
//   Vect out(vect.Data.size());

//   for (int row = 0; row < vect.Data.size(); ++row) {
//     out[row] = 0;
//     for (int i = 0; i < EntriesPerRow; i++) {
//       int col = floor(Indices.GetData(row, i).real());
//       th_cplx val = Values.GetData(row, i);

//       if (col > -1)
//         out.Data[row] += val * vect.Data[col];
//     }
//   }
//   return out;
// }
