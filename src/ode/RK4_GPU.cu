#include "hip/hip_runtime.h"
//
// Copyright (C) 2025 Conor Stevenson
// Licensed under the GNU General Public License v3.0
// Created by conor on 16/04/22.
//
#include "ode/RK4_GPU.cuh"
#include "ode/GPU.cuh"


RK4_GPU::RK4_GPU(th_hostVect &y0, double t0, SparseELL &M, double tol)
    : k1(y0.size(), 0), k2(y0.size(), 0),
      k3(y0.size(), 0), k4(y0.size(), 0)

{
    D_M_Values  = M.Values.FlattenedData();
    // D_M_Indices = M.Indices.FlattenedDataInt();
    n_columns   = M.EntriesPerRow;

    x = y0;
    thrust::copy(x.begin(), x.end(), D_x.begin());
}

void RK4_GPU::State(struct State &dst) {

}

double RK4_GPU::Step(double step) {
    th_cplx* D_x_ = thrust::raw_pointer_cast( D_x.data() );

    th_cplx* k1_ = thrust::raw_pointer_cast( k1.data() );
    th_cplx* k2_ = thrust::raw_pointer_cast( k2.data() );
    th_cplx* k3_ = thrust::raw_pointer_cast( k3.data() );
    th_cplx* k4_ = thrust::raw_pointer_cast( k4.data() );

    th_cplx* D_MValues_  = thrust::raw_pointer_cast( D_M_Values.data() );
    int*    D_MIndices_ = thrust::raw_pointer_cast( D_M_Indices.data() );

    return 0;
}

